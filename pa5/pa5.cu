#include "hip/hip_runtime.h"
//TJ Walker 1204996 and Jake Harwood 1226732
//assignment 5 comp sci 4F03


#include <stdlib.h>
#include <stdio.h>
extern "C" {
	#include "ppmFile.h"
}

#define MAX_NUM_ARGUMENTS 4

int blurRadius;
char * input_ppm_filename;
char * output_ppm_filename;
Image * inputPPM;
Image * outputPPM;

__device__ void blur(int x, int y, int blurRadius, int width, int height, unsigned char *input, unsigned char *output) {
	

	int minX = x - blurRadius;
	int minY = y - blurRadius;
	int maxX = x + blurRadius;
	int maxY = y + blurRadius;

	if (minX < 0) {
		minX = 0;
	}
	if (minY < 0) {
		minY = 0;
	}
	if (maxX >= width){
		maxX = width;
	}  
	if (maxY >= height){
		maxY = height;
	}

	unsigned long int rgbBLUR[3] = {0, 0, 0};

	for (int j = minY; j <= maxY; j++){
		for (int i = minX; i <= maxX; i++){
			for (int k = 0; k < 3; k++){
				rgbBLUR[k] += input[j * width * 3 + i * 3 + k];
				int numOfPixels = ((maxX - minX + 1) * (maxY - minY + 1));
				output[y * width * 3 + x * 3 + k] = (unsigned char)(rgbBLUR[k] / numOfPixels);
			}
		}
	}
}

__global__ void blurFilter(int width, int height, int blurRadius, unsigned char * inputImgData, unsigned char * outputImgData) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int	y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height){
		blur(x, y, blurRadius, width, height, inputImgData, outputImgData);
	}	
}



/* main */

int main(int argc, char **argv) {

	if (argc != MAX_NUM_ARGUMENTS) {
		printf("Too many arguments -- application quitting");
		exit(1);
	}

    blurRadius = atoi(argv[1]);
    input_ppm_filename= argv[2];
    output_ppm_filename = argv[3];
    inputPPM = ImageRead(input_ppm_filename);
	outputPPM = ImageCreate(inputPPM->width, inputPPM->height);
	int imageSize= inputPPM->width * inputPPM->height * 3;
	unsigned char *inputImgData, *outputImgData;

	int threadsPerBlock = 32;

	dim3 gridCustom(ceil((double)inputPPM->width / threadsPerBlock), ceil((double)inputPPM->height / threadsPerBlock), 1);
	dim3 blockCustom(threadsPerBlock, threadsPerBlock, 1);
	
	hipMalloc(&inputImgData, imageSize);
	hipMalloc(&outputImgData, imageSize);
	hipMemcpy(inputImgData, inputPPM->data, imageSize, hipMemcpyHostToDevice);

	blurFilter<<<gridCustom, blockCustom>>>(inputPPM->width, inputPPM->height, blurRadius, inputImgData, outputImgData);
	hipDeviceSynchronize();

	hipMemcpy(outputPPM->data, outputImgData, imageSize, hipMemcpyDeviceToHost);
	ImageWrite(outputPPM, argv[3]);

	hipFree(inputImgData);
	hipFree(outputImgData);
	
	free(inputPPM);
	free(outputPPM);

	return 0;
}


