#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(){
	int myID = (blockIdx.z * gridDim.x * gridDim.y +
				blockIdx.y * gridDim.x +
				blockIdx.x) * blockDim.x + threadIdx.x;
	printf("Hello World from thread %i \n", myID);
}

int main(){
	dim3 b(3,3,3); 
	hello<<<b,32>>>();
	hipDeviceSynchronize();
	return 1;

}